#include <hip/hip_runtime.h>


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/time.h>

#include <iostream>
#include <memory>
#include <cassert>

int __device__ min3(int a, int b, int c) {
    return ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)));
}


int __device__ levenshtein_d(char *s1, char *s2, int len, int *column) {
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++) {
        column[y] = y;
    }
    for (x = 1; x <= len; x++) {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++) {
            olddiag = column[y];
            column[y] = min3(
                    column[y] + 1,
                    column[y - 1] + 1,
                    lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1)
            );
            lastdiag = olddiag;

        }
    }
    return (column[len]);
}

void __global__ kernelwrapper(int* d_n_matches, char * d_buf, char * d_pattern, int i, int size_pattern, int offset, int n_bytes, int approx_factor){

    /* Traverse the input data up to the end of the file */
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int distance = 0 ;
    int size ;

    size = size_pattern ;
    int* columns = (int *) malloc((size_pattern + 1) * sizeof(int));
    while (j < n_bytes) {
        if (n_bytes - j < size_pattern ){
            size = n_bytes - j ;
        }

        distance = levenshtein_d(d_pattern + offset, &d_buf[j], size, columns ) ;
        if ( distance <= approx_factor) {
            atomicAdd(&d_n_matches[i], 1);
        }

        j += stride;
    }
    free(columns);

}



#define APM_DEBUG 0

char * 
read_input_file( char * filename, int * size )
{
    char * buf ;
    off_t fsize;
    int fd = 0 ;
    int n_bytes = 1 ;

    /* Open the text file */
    fd = open( filename, O_RDONLY ) ;
    if ( fd == -1 ) 
    {
        fprintf( stderr, "Unable to open the text file <%s>\n", filename ) ;
        return NULL ;
    }


    /* Get the number of characters in the textfile */
    fsize = lseek(fd, 0, SEEK_END);
    if ( fsize == -1 )
    {
        fprintf( stderr, "Unable to lseek to the end\n" ) ;
        return NULL ;
    }

#if APM_DEBUG
    printf( "File length: %lld\n", fsize ) ;
#endif

    /* Go back to the beginning of the input file */
    if ( lseek(fd, 0, SEEK_SET) == -1 ) 
    {
        fprintf( stderr, "Unable to lseek to start\n" ) ;
        return NULL ;
    }

    /* Allocate data to copy the target text */
    buf = (char *)malloc( fsize * sizeof ( char ) ) ;
    if ( buf == NULL ) 
    {
        fprintf( stderr, "Unable to allocate %lld byte(s) for main array\n",
                fsize ) ;
        return NULL ;
    }

    n_bytes = read( fd, buf, fsize ) ;
    if ( n_bytes != fsize ) 
    {
        fprintf( stderr, 
                "Unable to copy %lld byte(s) from text file (%d byte(s) copied)\n",
                fsize, n_bytes) ;
        return NULL ;
    }

#if APM_DEBUG
    printf( "Number of read bytes: %d\n", n_bytes ) ;
#endif

    *size = n_bytes ;


    close( fd ) ;


    return buf ;
}

int 
main( int argc, char ** argv )
{
  char ** pattern ;
  char * filename ;
  int approx_factor = 0 ;
  int nb_patterns = 0 ;
  int i, j ;
  char * buf ;
  struct timeval t1, t2, t3;
  double duration ;
  int n_bytes ;
  int * n_matches ;

  /* Check number of arguments */
  if ( argc < 4 ) 
  {
    printf( "Usage: %s approximation_factor "
            "dna_database pattern1 pattern2 ...\n", 
            argv[0] ) ;
    return 1 ;
  }

  /* Get the distance factor */
  approx_factor = atoi( argv[1] ) ;

  /* Grab the filename containing the target text */
  filename = argv[2] ;

  /* Get the number of patterns that the user wants to search for */
  nb_patterns = argc - 3 ;

  /* Fill the pattern array */
  pattern = (char **)malloc( nb_patterns * sizeof( char * ) ) ;
  if ( pattern == NULL ) 
  {
      fprintf( stderr, 
              "Unable to allocate array of pattern of size %d\n", 
              nb_patterns ) ;
      return 1 ;
  }

  /* Grab the patterns */
  for ( i = 0 ; i < nb_patterns ; i++ ) 
  {
      int l ;

      l = strlen(argv[i+3]) ;
      if ( l <= 0 ) 
      {
          fprintf( stderr, "Error while parsing argument %d\n", i+3 ) ;
          return 1 ;
      }

      pattern[i] = (char *)malloc( (l+1) * sizeof( char ) ) ;
      if ( pattern[i] == NULL ) 
      {
          fprintf( stderr, "Unable to allocate string of size %d\n", l ) ;
          return 1 ;
      }

      strncpy( pattern[i], argv[i+3], (l+1) ) ;
  }


  printf( "Approximate Pattern Mathing: "
          "looking for %d pattern(s) in file %s w/ distance of %d\n", 
          nb_patterns, filename, approx_factor ) ;

  buf = read_input_file( filename, &n_bytes ) ;
  if ( buf == NULL )
  {
      return 1 ;
  }

  /* Allocate the array of matches */
  n_matches = (int *)malloc( nb_patterns * sizeof( int ) ) ;
  if ( n_matches == NULL )
  {
      fprintf( stderr, "Error: unable to allocate memory for %ldB\n",
              nb_patterns * sizeof( int ) ) ;
      return 1 ;
  }


  /* Matching process takes place in GPU */

    int* d_n_matches;
    char * d_pattern;
    char* d_buf;
    int* offset = (int *)malloc( nb_patterns * sizeof( int ) ) ;
    int* lens = (int *)malloc( nb_patterns * sizeof( int ) ) ;
    int sum_lens;
    lens[0] = strlen(pattern[0]);
    offset[0] = 0;
    sum_lens = lens[0];
    for (i = 1; i < nb_patterns; i++) {
        offset[i] = offset[i-1] + lens[i-1];
        lens[i] = strlen(pattern[i]);
        sum_lens += lens[i];
    }
    char* concat_patterns = (char*) malloc( sum_lens * sizeof( char ) ) ;
    for (i = 0; i < nb_patterns; i++) {
        strcpy (concat_patterns + offset[i], pattern[i]);
    }
    
    gettimeofday(&t1, NULL);
    hipMalloc((void **)&d_n_matches, nb_patterns*sizeof(int));
    hipMalloc((void **)&d_pattern, sum_lens*sizeof(char));
    hipMalloc((void **)&d_buf, n_bytes);
    hipMemcpy(d_pattern, concat_patterns, sum_lens*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_buf, buf, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_n_matches, n_matches, nb_patterns*sizeof(int), hipMemcpyHostToDevice);

    gettimeofday(&t2, NULL);

    duration = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);

    printf( "Memory copy from host to device done in %lf s \n", duration) ;

    int Dg = 4;
    int Db = 256;
    for (i = 0; i < nb_patterns; i++) {
        kernelwrapper<<<Dg,Db>>>(d_n_matches, d_buf, d_pattern, i, lens[i], offset[i], n_bytes, approx_factor);
    }
    
    hipMemcpy(n_matches, d_n_matches, nb_patterns*sizeof(int), hipMemcpyDeviceToHost);

    gettimeofday(&t3, NULL);

    duration = (t3.tv_sec -t2.tv_sec)+((t3.tv_usec-t2.tv_usec)/1e6);

    printf( "Calculation on GPU done in %lf s \n", duration) ;


  for ( i = 0 ; i < nb_patterns ; i++ )
  {
      printf( "Number of matches for pattern <%s>: %d\n", 
              pattern[i], n_matches[i] ) ;
  }

  return 0 ;
}